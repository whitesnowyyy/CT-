#include "hip/hip_runtime.h"

#include <iostream>
#include <cmath>
#include <vector>
#include <hip/hip_runtime.h>

using namespace std;

#define PI 3.14159265358979323846
#define BLOCK_SIZE 16

global void compute_image_kernel(double* sinogram_gpu, double* image_gpu, double r, double pixel_s, double fr, double angel, double vs)
{
// 线程索引
int i = blockIdx.y * blockDim.y + threadIdx.y;
int j = blockIdx.x * blockDim.x + threadIdx.x;

// 如果超出像素范围则直接返回
if (i >= 400 || j >= 400)
    return;

double pa = 0;

for (int k = 0; k < 256; k++)
{
    double t = floor(sqrt(pow((j - 200) * pixel_s - cos(k * angel - PI) * r, 2) + pow((i - 200) * pixel_s - sin(k * angel - PI) * r, 2)) / (vs * fr)) + 1;
    pa = pa + sinogram_gpu[(int)t * 256 + k];
}

image_gpu[i * 400 + j] = pa;
}

void cuda_compute_image(std::vector<std::vector<double> > &sinogram, double *image, double r, double pixel_s, double fr, double angel, double vs)
{
// 声明GPU内存指针
double *sinogram_gpu, *image_gpu;

// 计算输入sinogram数组在设备内存中的尺寸
int sinogram_size = 4096 * 256 * sizeof(double);

// 计算输出image数组在设备内存中的尺寸
int image_size = 400 * 400 * sizeof(double);

// 在设备内存中分配输入与输出数组的空间
hipMalloc((void**)&sinogram_gpu, sinogram_size);
hipMalloc((void**)&image_gpu, image_size);

// 将输入sinogram数据复制到设备内存中
hipMemcpy(sinogram_gpu, &sinogram[0][0], sinogram_size, hipMemcpyHostToDevice);

// 定义线程块
dim3 block(BLOCK_SIZE, BLOCK_SIZE);
dim3 grid((400 + BLOCK_SIZE - 1) / BLOCK_SIZE, (400 + BLOCK_SIZE - 1) / BLOCK_SIZE);

// 执行GPU Kernel函数进行图像重建，并等待GPU处理完成
compute_image_kernel <<<grid, block >>> (sinogram_gpu, image_gpu, r, pixel_s, fr, angel, vs);
hipDeviceSynchronize();

// 将输出image数据从设备内存中复制回主机内存
hipMemcpy(image, image_gpu, image_size, hipMemcpyDeviceToHost);

// 释放GPU内存空间
hipFree(sinogram_gpu);
hipFree(image_gpu);
}

int main() {
std::vector<std::vector<double>> sinogram(4096, std::vector<double>(256, 1.0));

for (int i = 0; i < 2000; i++) {
    for (int j = 0; j < 256; j++) {
        sinogram[i][j] = 0.0;
    }
}

double image[400][400];

double r = 55e-3;
double pixel_s = 0.1e-3;
double fr = 40e6;
double angel = 2 * PI  / 256;
double vs = 1.5e3;

// 调用CUDA函数进行图像重建计算
cuda_compute_image(sinogram, &image[0][0], r, pixel_s, fr, angel, vs);

return 0;
}